#include "hip/hip_runtime.h"
#include <SFMLTexture.cuh>

SFMLTexture::SFMLTexture(Render3D *render) {
    texture.create(render->W_WIDTH, render->W_HEIGHT);
    pixels = new sf::Uint8[render->W_WIDTH * render->W_HEIGHT * 4];

    // Allocate memory for the Pixel buffer
    CUDA_CHECK(hipMalloc(&d_buffer, render->BUFFER_SIZE * sizeof(Pixel3D)));
    CUDA_CHECK(hipMalloc(&d_sfPixels, render->W_WIDTH * render->W_HEIGHT * 4));

    // Set kernel parameters
    numBlocks = (render->BUFFER_SIZE + blockSize - 1) / blockSize;

    // Create SFML sprite
    sprite = sf::Sprite(texture);
}
SFMLTexture::~SFMLTexture() {
    delete[] pixels;
    CUDA_CHECK(hipFree(d_buffer));
    CUDA_CHECK(hipFree(d_sfPixels));
}

void SFMLTexture::updateTexture(Render3D *render) {
    // Copy buffer to device
    CUDA_CHECK(hipMemcpy(d_buffer, render->BUFFER, render->BUFFER_SIZE * sizeof(Pixel3D), hipMemcpyHostToDevice));

    // Execute fillPixel kernel
    fillPixelKernel<<<numBlocks, blockSize>>>(
        d_sfPixels, d_buffer,
        render->BUFFER_WIDTH, render->BUFFER_HEIGHT,
        render->PIXEL_SIZE
    );
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy pixels back to host
    CUDA_CHECK(hipMemcpy(pixels, d_sfPixels, render->W_WIDTH * render->W_HEIGHT * 4, hipMemcpyDeviceToHost));

    // Update texture
    texture.update(pixels);
}

__global__ void fillPixelKernel(
    sf::Uint8 *pixels, Pixel3D *buffer,
    int b_w, int b_h, int p_s
) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < b_w * b_h) {
        int x = i % b_w;
        int y = i / b_w;
        int b_index = x + y * b_w;

        for (int i = 0; i < p_s; i++)
        for (int j = 0; j < p_s; j++) {
            int p_index = x * p_s + i + (y * p_s + j) * b_w * p_s;
            p_index *= 4;

            // Get the pixel color
            Color3D color = buffer[b_index].color;

            // Fill the pixel
            pixels[p_index] = color.runtimeRGB.v1;
            pixels[p_index + 1] = color.runtimeRGB.v2;
            pixels[p_index + 2] = color.runtimeRGB.v3;
            pixels[p_index + 3] = 255;
        }
    }
}