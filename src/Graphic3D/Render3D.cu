#include "hip/hip_runtime.h"
#include <Render3D.cuh>

#include <thrust/device_vector.h>
#include <thrust/sort.h>

Render3D::Render3D(Camera3D *camera, int w_w, int w_h, int p_s) {
    // Window settings
    W_WIDTH = w_w;
    W_HEIGHT = w_h;
    W_CENTER_X = w_w / 2;
    W_CENTER_Y = w_h / 2;
    PIXEL_SIZE = p_s;

    // Camera settings
    CAMERA = camera;
    CAMERA->w_width = w_w;
    CAMERA->w_height = w_h;
    CAMERA->w_center_x = W_CENTER_X;
    CAMERA->w_center_y = W_CENTER_Y;

    // Initialize buffer
    BUFFER_WIDTH = w_w / p_s;
    BUFFER_HEIGHT = w_h / p_s;
    BUFFER_SIZE = BUFFER_WIDTH * BUFFER_HEIGHT;
    BUFFER = new Pixel3D[BUFFER_SIZE];

    // Memory allocation for device buffer
    CUDA_CHECK(hipMalloc(&D_BUFFER, BUFFER_SIZE * sizeof(Pixel3D)));
}
Render3D::~Render3D() {
    delete[] BUFFER;
    CUDA_CHECK(hipFree(D_BUFFER));
}

void Render3D::resize(int w_w, int w_h, int p_s) {
    // Update window settings
    W_WIDTH = w_w;
    W_HEIGHT = w_h;
    W_CENTER_X = w_w / 2;
    W_CENTER_Y = w_h / 2;
    PIXEL_SIZE = p_s;

    // Update buffer settings
    BUFFER_WIDTH = w_w / p_s;
    BUFFER_HEIGHT = w_h / p_s;
    BUFFER_SIZE = BUFFER_WIDTH * BUFFER_HEIGHT;

    // Reset buffer
    delete[] BUFFER;
    BUFFER = new Pixel3D[BUFFER_SIZE];

    // Free old device memory
    CUDA_CHECK(hipFree(D_BUFFER));
    // Memory allocation for device buffer
    CUDA_CHECK(hipMalloc(&D_BUFFER, BUFFER_SIZE * sizeof(Pixel3D)));
}

// Reset all
void Render3D::reset() {
    // Reset buffer
    delete[] BUFFER;
    BUFFER = new Pixel3D[BUFFER_SIZE];
}

// To vec2D
__host__ __device__ Vec2D Render3D::toVec2D(const Camera3D &cam, Vec3D v) {
    Vec3D diff = Vec3D::sub(v, cam.pos);

    // Apply Yaw (rotation around Y axis)
    float cosYaw = cos(-cam.ang.y);
    float sinYaw = sin(-cam.ang.y);
    float tempX = diff.x * cosYaw + diff.z * sinYaw;
    float tempZ = -diff.x * sinYaw + diff.z * cosYaw;

    // Apply Pitch (rotation around X axis)
    float cosPitch = cos(-cam.ang.x);
    float sinPitch = sin(-cam.ang.x);
    float finalY = tempZ * sinPitch + diff.y * cosPitch;
    float finalZ = tempZ * cosPitch - diff.y * sinPitch;

    float projX = (tempX * cam.screendist) / finalZ;
    float projY = -(finalY * cam.screendist) / finalZ;

    if (finalZ < 0) {
        projX *= -10;
        projY *= -10;
    }

    projX += cam.w_center_x;
    projY += cam.w_center_y;

    return Vec2D(projX, projY, finalZ);
}

// The main render function
void Render3D::renderGPU(Tri3D *tri3Ds, size_t size) {
    // Set kernel parameters
    const size_t numBlocks = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Allocate triangles memory on device
    CUDA_CHECK(hipMalloc(&D_TRI3DS, size * sizeof(Tri3D)));
    CUDA_CHECK(hipMalloc(&D_TRI2DS, size * sizeof(Tri2D)));

    // Copy data to device
    CUDA_CHECK(hipMemcpy(D_TRI3DS, tri3Ds, size * sizeof(Tri3D), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(D_BUFFER, BUFFER, BUFFER_SIZE * sizeof(Pixel3D), hipMemcpyHostToDevice));

    // Execute tri3DsTo2Ds kernel
    tri3DsTo2DsKernel<<<numBlocks, BLOCK_SIZE>>>(
        D_TRI2DS, D_TRI3DS, *CAMERA, PIXEL_SIZE, size
    );
    CUDA_CHECK(hipDeviceSynchronize());

    // Sort the 2D triangles by zDepth (and rearrange the 3D triangles accordingly)
    thrust::device_vector<Tri2D> dev_tri2Ds(D_TRI2DS, D_TRI2DS + size);
    thrust::device_vector<Tri3D> dev_tri3Ds(D_TRI3DS, D_TRI3DS + size);
    // Sort using the thrust, while also rearranging the tri3Ds
    thrust::sort_by_key(dev_tri2Ds.begin(), dev_tri2Ds.end(), dev_tri3Ds.begin(),
        [] __device__ (const Tri2D& a, const Tri2D& b) -> bool {
            return a.v1.zDepth > b.v1.zDepth;
        }
    );
    // Copy back to device memory
    thrust::copy(dev_tri2Ds.begin(), dev_tri2Ds.end(), D_TRI2DS);
    thrust::copy(dev_tri3Ds.begin(), dev_tri3Ds.end(), D_TRI3DS);

    // Execute rasterization kernel
    rasterizeKernel<<<numBlocks, BLOCK_SIZE>>>(
        // Buffer and tris
        D_BUFFER, D_TRI2DS, D_TRI3DS,
        // Other properties if needed
        LIGHT,
        // Size properties
        BUFFER_WIDTH, BUFFER_HEIGHT, size
    );
    CUDA_CHECK(hipDeviceSynchronize());

    // For every unoccupied pixel, fill it with the default color
    fillBufferKernel<<<BUFFER_SIZE / BLOCK_SIZE + 1, BLOCK_SIZE>>>(
        D_BUFFER, DEFAULT_COLOR, BUFFER_SIZE
    );
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy pixels back to host buffer
    CUDA_CHECK(hipMemcpy(BUFFER, D_BUFFER, BUFFER_SIZE * sizeof(Pixel3D), hipMemcpyDeviceToHost));

    // Free device memory
    CUDA_CHECK(hipFree(D_TRI3DS));
    CUDA_CHECK(hipFree(D_TRI2DS));
}

void Render3D::renderCPU(std::vector<Tri3D> tri3Ds) {
    // Decrapitated
}

__global__ void fillBufferKernel(
    Pixel3D *buffer, Color3D color, size_t size
) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size && !buffer[i].active) buffer[i].color = color;
}

__global__ void tri3DsTo2DsKernel(
    Tri2D *tri2Ds, const Tri3D *tri3Ds,
    Camera3D cam, int p_s, size_t size
) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        Vec2D v1 = Render3D::toVec2D(cam, tri3Ds[i].v1);
        Vec2D v2 = Render3D::toVec2D(cam, tri3Ds[i].v2);
        Vec2D v3 = Render3D::toVec2D(cam, tri3Ds[i].v3);

        // Divide by pixel size
        v1.x /= p_s; v1.y /= p_s;
        v2.x /= p_s; v2.y /= p_s;
        v3.x /= p_s; v3.y /= p_s;

        // IMPORTANT: v1 -> v3 will have ascending zDepth
        // (note: we cannot use std::swap in device code)
        tri2Ds[i].v1 = v1;
        tri2Ds[i].v2 = v2;
        tri2Ds[i].v3 = v3;
    }
}

__device__ bool atomicMinFloat(float* addr, float value) {
    int* addr_as_int = (int*)addr;
    int old = *addr_as_int, assumed;

    do {
        assumed = old;
        old = atomicCAS(addr_as_int, assumed, __float_as_int(fminf(value, __int_as_float(assumed))));
    } while (assumed != old);

    return __int_as_float(old) > value;
}

__global__ void rasterizeKernel(
    // Buffer and tris
    Pixel3D *pixels, const Tri2D *tri2Ds, const Tri3D *tri3Ds,
    // Other properties if needed
    LightSrc3D light,
    // Size properties
    int b_w, int b_h, size_t size
) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        // If the triangle is not visible, skip
        if (tri2Ds[i].v1.zDepth < 0 && tri2Ds[i].v2.zDepth < 0 && tri2Ds[i].v3.zDepth < 0) return;
        if (tri2Ds[i].v1.x < 0 && tri2Ds[i].v2.x < 0 && tri2Ds[i].v3.x < 0) return;
        if (tri2Ds[i].v1.y < 0 && tri2Ds[i].v2.y < 0 && tri2Ds[i].v3.y < 0) return;
        if (tri2Ds[i].v1.x >= b_w && tri2Ds[i].v2.x >= b_w && tri2Ds[i].v3.x >= b_w) return;
        if (tri2Ds[i].v1.y >= b_h && tri2Ds[i].v2.y >= b_h && tri2Ds[i].v3.y >= b_h) return;

        // Find the bounding box of the 2D polygon
        int minX = min(tri2Ds[i].v1.x, min(tri2Ds[i].v2.x, tri2Ds[i].v3.x));
        int maxX = max(tri2Ds[i].v1.x, max(tri2Ds[i].v2.x, tri2Ds[i].v3.x));
        int minY = min(tri2Ds[i].v1.y, min(tri2Ds[i].v2.y, tri2Ds[i].v3.y));
        int maxY = max(tri2Ds[i].v1.y, max(tri2Ds[i].v2.y, tri2Ds[i].v3.y));

        // Clip the bounding box (slightly expanded)
        minX = max(minX, 1);
        maxX = min(maxX, b_w - 2);
        minY = max(minY, 1);
        maxY = min(maxY, b_h - 2);

        // Rasterize the triangle using the baricentric coordinates
        for (int x = minX - 1; x <= maxX + 1; x++)
        for (int y = minY - 1; y <= maxY + 1; y++) {
            int index = x + y * b_w;
            // Check if the pixel is inside the triangle
            Vec2D screen(x, y);

            Vec3D barycentric = Vec2D::barycentricLambda(
                screen, tri2Ds[i].v1, tri2Ds[i].v2, tri2Ds[i].v3
            );

            // Check if the pixel is inside the triangle
            // (allow small margin of error)
            if (barycentric.x < 0.0 ||
                barycentric.y < 0.0 ||
                barycentric.z < 0.0) continue;

            screen.zDepth = Vec2D::barycentricCalc(
                barycentric, tri2Ds[i].v1.zDepth, tri2Ds[i].v2.zDepth, tri2Ds[i].v3.zDepth
            );

            // Check if the pixel is closer than the current pixel
            if (!atomicMinFloat(&pixels[index].screen.zDepth, screen.zDepth))
                continue;

            // Get world position
            double px = Vec2D::barycentricCalc(
                barycentric, tri3Ds[i].v1.x, tri3Ds[i].v2.x, tri3Ds[i].v3.x
            );
            double py = Vec2D::barycentricCalc(
                barycentric, tri3Ds[i].v1.y, tri3Ds[i].v2.y, tri3Ds[i].v3.y
            );
            double pz = Vec2D::barycentricCalc(
                barycentric, tri3Ds[i].v1.z, tri3Ds[i].v2.z, tri3Ds[i].v3.z
            );
            Vec3D world(px, py, pz);

            // BETA: Light color manipulation
            Color3D color = tri3Ds[i].color;

            Vec3D lightDir = Vec3D::sub(light.pos, world);
            double cosA = Vec3D::dot(tri3Ds[i].normal, lightDir) /
                (Vec3D::mag(tri3Ds[i].normal) * Vec3D::mag(lightDir));
            // Note: we cannot use std::max and std::min in device code
            if (cosA < 0) cosA = 0;
            // if (cosA < 0) cosA = -cosA;

            double ratio = light.ambient + cosA * (light.specular - light.ambient);

            color.runtimeRGB.mult(ratio);

            // Apply colored light
            color.runtimeRGB.v1 = color.runtimeRGB.v1 * light.rgbRatio.x;
            color.runtimeRGB.v2 = color.runtimeRGB.v2 * light.rgbRatio.y;
            color.runtimeRGB.v3 = color.runtimeRGB.v3 * light.rgbRatio.z;

            // Restrict color values
            color.runtimeRGB.restrictRGB();

            // Set buffer values
            pixels[index] = {
                color, tri3Ds[i].normal, world, screen, true
            };
        }
    }
}