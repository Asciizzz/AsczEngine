#include "hip/hip_runtime.h"
#include <Render3D.cuh>

#include <thrust/device_vector.h>
#include <thrust/sort.h>

Render3D::Render3D(Camera3D *camera) {
    this->camera = camera;

    // Initialize buffer
    BUFFER_WIDTH = W_WIDTH / PIXEL_SIZE;
    BUFFER_HEIGHT = W_HEIGHT / PIXEL_SIZE;
    BUFFER_SIZE = BUFFER_WIDTH * BUFFER_HEIGHT;
    BUFFER = new Pixel3D[BUFFER_SIZE];

    // Memory allocation for device buffer
    CUDA_CHECK(hipMalloc(&D_BUFFER, BUFFER_SIZE * sizeof(Pixel3D)));
}
Render3D::~Render3D() {
    delete[] BUFFER;

    // Free device memory
    CUDA_CHECK(hipFree(D_BUFFER));
}

// Reset all
void Render3D::reset() {
    // Reset buffer
    delete[] BUFFER;
    BUFFER = new Pixel3D[BUFFER_SIZE];
}

// To vec2D
__host__ __device__ Vec2D Render3D::toVec2D(const Camera3D &cam, Vec3D v) {
    Vec3D diff = Vec3D::sub(v, cam.pos);

    // Apply Yaw (rotation around Y axis)
    float cosYaw = cos(-cam.ang.y);
    float sinYaw = sin(-cam.ang.y);
    float tempX = diff.x * cosYaw + diff.z * sinYaw;
    float tempZ = -diff.x * sinYaw + diff.z * cosYaw;

    // Apply Pitch (rotation around X axis)
    float cosPitch = cos(-cam.ang.x);
    float sinPitch = sin(-cam.ang.x);
    float finalY = tempZ * sinPitch + diff.y * cosPitch;
    float finalZ = tempZ * cosPitch - diff.y * sinPitch;

    float projX = (tempX * cam.screendist) / finalZ;
    float projY = -(finalY * cam.screendist) / finalZ;

    if (finalZ < 0) {
        projX *= -10;
        projY *= -10;
    }

    projX += cam.w_center_x;
    projY += cam.w_center_y;

    return Vec2D(projX, projY, finalZ);
}

// The main render function
void Render3D::renderGPU(Tri3D *tri3Ds, size_t size) {
    // Set kernel parameters
    const size_t blockSize = 256;
    const size_t numBlocks = (size + blockSize - 1) / blockSize;

    // Allocate triangles memory on device
    CUDA_CHECK(hipMalloc(&D_TRI3DS, size * sizeof(Tri3D)));
    CUDA_CHECK(hipMalloc(&D_TRI2DS, size * sizeof(Tri2D)));

    // Copy data to device
    CUDA_CHECK(hipMemcpy(D_TRI3DS, tri3Ds, size * sizeof(Tri3D), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(D_BUFFER, BUFFER, BUFFER_SIZE * sizeof(Pixel3D), hipMemcpyHostToDevice));

    // Execute tri3DsTo2Ds kernel
    tri3DsTo2DsKernel<<<numBlocks, blockSize>>>(
        D_TRI2DS, D_TRI3DS, *camera, PIXEL_SIZE, size
    );
    CUDA_CHECK(hipDeviceSynchronize());

    // Sort the 2D triangles by zDepth (and rearrange the 3D triangles accordingly)
    thrust::device_vector<Tri2D> dev_tri2Ds(D_TRI2DS, D_TRI2DS + size);
    thrust::device_vector<Tri3D> dev_tri3Ds(D_TRI3DS, D_TRI3DS + size);
    // Sort using the thrust, while also rearranging the tri3Ds
    thrust::sort_by_key(dev_tri2Ds.begin(), dev_tri2Ds.end(), dev_tri3Ds.begin(),
        [] __device__ (const Tri2D& a, const Tri2D& b) -> bool {
            return a.v1.zDepth > b.v1.zDepth;
        }
    );
    // Copy back to device memory
    thrust::copy(dev_tri2Ds.begin(), dev_tri2Ds.end(), D_TRI2DS);
    thrust::copy(dev_tri3Ds.begin(), dev_tri3Ds.end(), D_TRI3DS);

    // Execute rasterization kernel
    rasterizeKernel<<<numBlocks, blockSize>>>(
        // Buffer and tris
        D_BUFFER, D_TRI2DS, D_TRI3DS,
        // Other properties if needed
        light,
        // Size properties
        BUFFER_WIDTH, BUFFER_HEIGHT, size
    );
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy pixels back to host buffer
    CUDA_CHECK(hipMemcpy(BUFFER, D_BUFFER, BUFFER_SIZE * sizeof(Pixel3D), hipMemcpyDeviceToHost));

    // Free device memory
    CUDA_CHECK(hipFree(D_TRI3DS));
    CUDA_CHECK(hipFree(D_TRI2DS));
}

void Render3D::renderCPU(std::vector<Tri3D> tri3Ds) {
    // Decrapitated
}

// HOLY SHIT THIS REDUCES THE RACE CONDITION BY ALOT
/* Explaination:

Parallelize rasterization on a singular buffer can 
*/
__device__ inline bool atomicMinDouble(double *address, double val) {
    unsigned long long int *address_as_ull = (unsigned long long int *) address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        if (__longlong_as_double(assumed) <= val) break;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val));
    } while (assumed != old);

    return __longlong_as_double(old) > val;
}

__global__ void tri3DsTo2DsKernel(
    Tri2D *tri2Ds, const Tri3D *tri3Ds, Camera3D cam, int p_s, size_t size
) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        Vec2D v1 = Render3D::toVec2D(cam, tri3Ds[i].v1);
        Vec2D v2 = Render3D::toVec2D(cam, tri3Ds[i].v2);
        Vec2D v3 = Render3D::toVec2D(cam, tri3Ds[i].v3);

        // Divide by pixel size
        v1.x /= p_s; v1.y /= p_s;
        v2.x /= p_s; v2.y /= p_s;
        v3.x /= p_s; v3.y /= p_s;

        // IMPORTANT: v1 -> v3 will have ascending zDepth
        // (note: we cannot use std::swap in device code)

        tri2Ds[i].v1 = v1;
        tri2Ds[i].v2 = v2;
        tri2Ds[i].v3 = v3;
    }
}

__global__ void rasterizeKernel(
    // Buffer and tris
    Pixel3D *pixels, const Tri2D *tri2Ds, const Tri3D *tri3Ds,
    // Other properties if needed
    LightSrc3D light,
    // Size properties
    int b_w, int b_h, size_t size
) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        // If the largest Z is less than 0, then do nothing
        if (tri2Ds[i].v3.zDepth < 0) return;

        // If all 3 x or y are out of bounds, then do nothing
        if (tri2Ds[i].v1.x < 0 && tri2Ds[i].v2.x < 0 && tri2Ds[i].v3.x < 0) return;
        if (tri2Ds[i].v1.y < 0 && tri2Ds[i].v2.y < 0 && tri2Ds[i].v3.y < 0) return;
        if (tri2Ds[i].v1.x >= b_w && tri2Ds[i].v2.x >= b_w && tri2Ds[i].v3.x >= b_w) return;
        if (tri2Ds[i].v1.y >= b_h && tri2Ds[i].v2.y >= b_h && tri2Ds[i].v3.y >= b_h) return;

        // Find the bounding box of the 2D polygon
        int minX = min(tri2Ds[i].v1.x, min(tri2Ds[i].v2.x, tri2Ds[i].v3.x));
        int maxX = max(tri2Ds[i].v1.x, max(tri2Ds[i].v2.x, tri2Ds[i].v3.x));
        int minY = min(tri2Ds[i].v1.y, min(tri2Ds[i].v2.y, tri2Ds[i].v3.y));
        int maxY = max(tri2Ds[i].v1.y, max(tri2Ds[i].v2.y, tri2Ds[i].v3.y));

        // Clip the bounding box (slightly expanded)
        minX = max(minX, 1);
        maxX = min(maxX, b_w - 2);
        minY = max(minY, 1);
        maxY = min(maxY, b_h - 2);

        // Rasterize the triangle using the baricentric coordinates
        for (int x = minX - 1; x <= maxX + 1; x++)
        for (int y = minY - 1; y <= maxY + 1; y++) {
            int index = x + y * b_w;
            // Check if the pixel is inside the triangle
            Vec2D p(x, y);

            Vec3D barycentric = Vec2D::barycentricLambda(
                p, tri2Ds[i].v1, tri2Ds[i].v2, tri2Ds[i].v3
            );

            // Check if the pixel is inside the triangle
            // (allow small margin of error)
            if (barycentric.x < 0.0 ||
                barycentric.y < 0.0 ||
                barycentric.z < 0.0) continue;

            p.zDepth = Vec2D::barycentricCalc(
                barycentric, tri2Ds[i].v1.zDepth, tri2Ds[i].v2.zDepth, tri2Ds[i].v3.zDepth
            );

            // Check if the pixel is closer than the current pixel
            if (!atomicMinDouble(&pixels[index].screen.zDepth, p.zDepth)) continue;
            // if (pixels[index].screen.zDepth < p.zDepth) continue;

            // Get world position
            double px = Vec2D::barycentricCalc(
                barycentric, tri3Ds[i].v1.x, tri3Ds[i].v2.x, tri3Ds[i].v3.x
            );
            double py = Vec2D::barycentricCalc(
                barycentric, tri3Ds[i].v1.y, tri3Ds[i].v2.y, tri3Ds[i].v3.y
            );
            double pz = Vec2D::barycentricCalc(
                barycentric, tri3Ds[i].v1.z, tri3Ds[i].v2.z, tri3Ds[i].v3.z
            );
            Vec3D worldPos(px, py, pz);

            // BETA: Light color manipulation
            Color3D color = tri3Ds[i].color;

            Vec3D lightDir = Vec3D::sub(light.pos, worldPos);

            double cosA = Vec3D::dot(tri3Ds[i].normal, lightDir) /
                (Vec3D::mag(tri3Ds[i].normal) * Vec3D::mag(lightDir));
            // Note: we cannot use std::max and std::min in device code
            if (cosA < 0) cosA = 0;

            double ratio = light.ambient + cosA * (light.specular - light.ambient);
            color.runtimeRGB = Color3D::x255(color.rawRGB);

            color.runtimeRGB.mult(ratio);

            // Apply colored light
            color.runtimeRGB.v1 = color.runtimeRGB.v1 * light.rgbRatio.x;
            color.runtimeRGB.v2 = color.runtimeRGB.v2 * light.rgbRatio.y;
            color.runtimeRGB.v3 = color.runtimeRGB.v3 * light.rgbRatio.z;

            // Restrict color values
            color.runtimeRGB.restrict(true);

            // Set buffer values
            pixels[index] = {
                color, tri3Ds[i].normal, worldPos, p
            };
        }
    }
}