#include "hip/hip_runtime.h"
#include <Render3D.cuh>

Render3D::Render3D(Camera3D *camera) {
    this->camera = camera;

    // Initialize buffer
    BUFFER_WIDTH = W_WIDTH / PIXEL_SIZE;
    BUFFER_HEIGHT = W_HEIGHT / PIXEL_SIZE;
    BUFFER_SIZE = BUFFER_WIDTH * BUFFER_HEIGHT;
    BUFFER = new Pixel3D[BUFFER_SIZE];

    // Memory allocation for device buffer
    CUDA_CHECK(hipMalloc(&D_BUFFER, BUFFER_SIZE * sizeof(Pixel3D)));

    // Memory allocation for device triangles will be done in renderGPU
    // As the count is dynamic and will be different each time
}
Render3D::~Render3D() {
    delete[] BUFFER;

    // Free device memory
    CUDA_CHECK(hipFree(D_BUFFER));
    CUDA_CHECK(hipFree(D_TRI3DS));
    CUDA_CHECK(hipFree(D_TRI2DS));
}

// Reset all
void Render3D::reset() {
    // Reset buffer
    delete[] BUFFER;
    BUFFER = new Pixel3D[BUFFER_SIZE];
}

// To vec2D
__host__ __device__ Vec2D Render3D::toVec2D(const Camera3D &cam, Vec3D v) {
    Vec3D diff = Vec3D::sub(v, cam.pos);

    // Rotation around Y-axis
    double transX = diff.x * cos(-cam.ang.y) + diff.z * sin(-cam.ang.y);
    double transZY = diff.z * cos(-cam.ang.y) - diff.x * sin(-cam.ang.y);

    // Rotation around X-axis
    double transY = diff.y * cos(-cam.ang.x) + transZY * sin(-cam.ang.x);
    double transZ = transZY * cos(-cam.ang.x) - diff.y * sin(-cam.ang.x);

    Vec2D vertex2D = {
        transX * cam.screendist / transZ,
        -transY * cam.screendist / transZ, // minus to flip
        transZ
    };

    if (vertex2D.zDepth <= 0) {
        vertex2D.x *= -10;
        vertex2D.y *= -10;
    }

    vertex2D.x += cam.w_center_x;
    vertex2D.y += cam.w_center_y;

    return vertex2D;
}

// The main render function
void Render3D::renderGPU(Tri3D *tri3Ds, size_t size) {
    Tri2D *tri2Ds = new Tri2D[size];

    // Set kernel parameters
    const size_t blockSize = 256;
    const size_t numBlocks = (size + blockSize - 1) / blockSize;

    // Allocate triangles memory on device
    CUDA_CHECK(hipMalloc(&D_TRI3DS, size * sizeof(Tri3D)));
    CUDA_CHECK(hipMalloc(&D_TRI2DS, size * sizeof(Tri2D)));

    // Copy data to device
    CUDA_CHECK(hipMemcpy(D_TRI3DS, tri3Ds, size * sizeof(Tri3D), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(D_BUFFER, BUFFER, BUFFER_SIZE * sizeof(Pixel3D), hipMemcpyHostToDevice));

    // Execute tri3DsTo2Ds kernel
    tri3DsTo2DsKernel<<<numBlocks, blockSize>>>(
        D_TRI2DS, D_TRI3DS, *camera, PIXEL_SIZE, size
    );
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy tri2Ds back to host
    CUDA_CHECK(hipMemcpy(tri2Ds, D_TRI2DS, size * sizeof(Tri2D), hipMemcpyDeviceToHost));

    // Execute rasterization kernel
    rasterizeKernel<<<numBlocks, blockSize>>>(
        // Buffer and tris
        D_BUFFER, D_TRI2DS, D_TRI3DS,
        // Other properties if needed
        light,
        // Size properties
        BUFFER_WIDTH, BUFFER_HEIGHT, size
    );
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy pixels back to host buffer
    CUDA_CHECK(hipMemcpy(BUFFER, D_BUFFER, BUFFER_SIZE * sizeof(Pixel3D), hipMemcpyDeviceToHost));

    // Free tri2Ds host memory
    delete[] tri2Ds;

    // Free device memory
    CUDA_CHECK(hipFree(D_TRI3DS));
    CUDA_CHECK(hipFree(D_TRI2DS));
}

void Render3D::renderCPU(std::vector<Tri3D> tri3Ds) {
    // Decrapitated
}

// KERNER FOR TRIANGLE RENDERING

/* Idea:

n 3D TRIs --/Parallel/--> n 2D TRIs + 1 buffer<> --/Parallel/--> 1 buffer<Pixels>

*/

__global__ void tri3DsTo2DsKernel(
    Tri2D *tri2Ds, const Tri3D *tri3Ds, Camera3D cam, int p_s, size_t size
) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        Vec2D v1 = Render3D::toVec2D(cam, tri3Ds[i].v1);
        Vec2D v2 = Render3D::toVec2D(cam, tri3Ds[i].v2);
        Vec2D v3 = Render3D::toVec2D(cam, tri3Ds[i].v3);

        // Divide by pixel size
        v1.x /= p_s; v1.y /= p_s;
        v2.x /= p_s; v2.y /= p_s;
        v3.x /= p_s; v3.y /= p_s;

        tri2Ds[i].v1 = v1;
        tri2Ds[i].v2 = v2;
        tri2Ds[i].v3 = v3;
    }
}

__global__ void rasterizeKernel(
    // Buffer and tris
    Pixel3D *pixels, const Tri2D *tri2Ds, const Tri3D *tri3Ds,
    // Other properties if needed
    LightSrc3D light,
    // Size properties
    int b_w, int b_h, size_t size
) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        // If all 3 zDepth are negative, then then do nothing
        if (tri2Ds[i].v1.zDepth <= 0 &&
            tri2Ds[i].v2.zDepth <= 0 &&
            tri2Ds[i].v3.zDepth <= 0) return;

        // Find the bounding box of the 2D polygon
        int minX = min(tri2Ds[i].v1.x, min(tri2Ds[i].v2.x, tri2Ds[i].v3.x));
        int maxX = max(tri2Ds[i].v1.x, max(tri2Ds[i].v2.x, tri2Ds[i].v3.x));
        int minY = min(tri2Ds[i].v1.y, min(tri2Ds[i].v2.y, tri2Ds[i].v3.y));
        int maxY = max(tri2Ds[i].v1.y, max(tri2Ds[i].v2.y, tri2Ds[i].v3.y));

        // Clip the bounding box (slightly expanded)
        minX = max(minX, 1);
        maxX = min(maxX, b_w - 2);
        minY = max(minY, 1);
        maxY = min(maxY, b_h - 2);

        // Rasterize the triangle using the baricentric coordinates
        for (int x = minX - 1; x <= maxX + 1; x++)
        for (int y = minY - 1; y <= maxY + 1; y++) {
            int index = x + y * b_w;
            // Check if the pixel is inside the triangle
            Vec2D p(x, y);

            Vec3D barycentric = Vec2D::barycentricLambda(
                p, tri2Ds[i].v1, tri2Ds[i].v2, tri2Ds[i].v3
            );

            // Check if the pixel is inside the triangle
            // (allow small margin of error)
            if (barycentric.x < -0.01 ||
                barycentric.y < -0.01 ||
                barycentric.z < -0.01) continue;

            p.zDepth = Vec2D::barycentricCalc(
                barycentric, tri2Ds[i].v1.zDepth, tri2Ds[i].v2.zDepth, tri2Ds[i].v3.zDepth
            );

            // Check if the pixel is closer than the current pixel
            if (pixels[index].screen.zDepth < p.zDepth) continue;

            // Get world position
            double px = Vec2D::barycentricCalc(
                barycentric, tri3Ds[i].v1.x, tri3Ds[i].v2.x, tri3Ds[i].v3.x
            );
            double py = Vec2D::barycentricCalc(
                barycentric, tri3Ds[i].v1.y, tri3Ds[i].v2.y, tri3Ds[i].v3.y
            );
            double pz = Vec2D::barycentricCalc(
                barycentric, tri3Ds[i].v1.z, tri3Ds[i].v2.z, tri3Ds[i].v3.z
            );
            Vec3D worldPos(px, py, pz);

            // BETA: Light color manipulation
            Color3D color = tri3Ds[i].color;

            Vec3D lightDir = Vec3D::sub(light.pos, worldPos);

            double cosA = Vec3D::dot(tri3Ds[i].normal, lightDir) /
                (Vec3D::mag(tri3Ds[i].normal) * Vec3D::mag(lightDir));
            // Note: we cannot use std::max and std::min in device code
            if (cosA < 0) cosA = 0;

            double ratio = light.ambient + cosA * (light.specular - light.ambient);
            color.runtimeRGB = Color3D::x255(color.rawRGB);

            color.runtimeRGB.v1 = color.runtimeRGB.v1 * ratio;
            color.runtimeRGB.v2 = color.runtimeRGB.v2 * ratio;
            color.runtimeRGB.v3 = color.runtimeRGB.v3 * ratio;

            // Apply colored light
            color.runtimeRGB.v1 = color.runtimeRGB.v1 * light.rgbRatio.x;
            color.runtimeRGB.v2 = color.runtimeRGB.v2 * light.rgbRatio.y;
            color.runtimeRGB.v3 = color.runtimeRGB.v3 * light.rgbRatio.z;

            // Restrict color values
            if (color.runtimeRGB.v1 > 255) color.runtimeRGB.v1 = 255;
            if (color.runtimeRGB.v2 > 255) color.runtimeRGB.v2 = 255;
            if (color.runtimeRGB.v3 > 255) color.runtimeRGB.v3 = 255;

            if (color.runtimeRGB.v1 < 0) color.runtimeRGB.v1 = 0;
            if (color.runtimeRGB.v2 < 0) color.runtimeRGB.v2 = 0;
            if (color.runtimeRGB.v3 < 0) color.runtimeRGB.v3 = 0;

            // Set buffer values
            pixels[index] = {
                color, tri3Ds[i].normal, worldPos, p
            };
        }
    }
}