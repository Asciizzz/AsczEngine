#include "hip/hip_runtime.h"
#include <Color3D.cuh>

// Custom functions that replace some library functions
#include <iostream>

// Custom implementation of fmod
__host__ __device__ double custom_fmod(double x, double y) {
    // Handle case where y is 0 to avoid division by zero
    if (y == 0.0) {
        return x;  // or some other appropriate value/behavior
    }

    double quotient = x / y;
    // Use truncation to get the integer part
    double integer_part;
    
    // Getting the integer part by truncating the quotient
    if (quotient >= 0) {
        integer_part = static_cast<double>(static_cast<int>(quotient));
    } else {
        integer_part = static_cast<double>(static_cast<int>(quotient) - 1);
    }
    
    // Return the remainder
    return x - (integer_part * y);
}

// ColorVec

__host__ __device__ void ColorVec::mult(double scalar) {
    v1 *= scalar;
    v2 *= scalar;
    v3 *= scalar;
}
__host__ __device__ void ColorVec::restrictRGB() {
    v1 = std::min(255.0, std::max(0.0, v1));
    v2 = std::min(255.0, std::max(0.0, v2));
    v3 = std::min(255.0, std::max(0.0, v3));
}

__host__ __device__ Color3D::Color3D(double r, double g, double b, double a) {
    // RGB
    rawRGB = {r, g, b};
    runtimeRGB = rawRGB;

    // HSL
    rawHSL = toHSL(rawRGB);
    runtimeHSL = rawHSL;

    // Alpha
    alpha = a;
}

__host__ __device__ ColorVec Color3D::toHSL(ColorVec rgb) {
    double r = rgb.v1 / 255;
    double g = rgb.v2 / 255;
    double b = rgb.v3 / 255;

    double max = std::max(r, std::max(g, b));
    double min = std::min(r, std::min(g, b));
    double h, s, l = (max + min) / 2;

    if (max == min) {
        h = s = 0; // achromatic
    } else {
        double d = max - min;
        s = l > 0.5 ? d / (2 - max - min) : d / (max + min);

        if (max == r) h = (g - b) / d + (g < b ? 6 : 0);
        else if (max == g) h = (b - r) / d + 2;
        else if (max == b) h = (r - g) / d + 4;

        h /= 6;
    }

    return {h, s, l};
}

__host__ __device__ ColorVec Color3D::toRGB(ColorVec hsl) {
    double h = hsl.v1;
    double s = hsl.v2;
    double l = hsl.v3;

    double r, g, b;

    double c = (1 - std::abs(2 * l - 1)) * s;
    double x = c * (1 - std::abs(custom_fmod(h * 6, 2) - 1));
    double m = l - c / 2;

    if (h < 1.0 / 6) { r = c; g = x; b = 0; }
    else if (h < 2.0 / 6) { r = x; g = c; b = 0; }
    else if (h < 3.0 / 6) { r = 0; g = c; b = x; }
    else if (h < 4.0 / 6) { r = 0; g = x; b = c; }
    else if (h < 5.0 / 6) { r = x; g = 0; b = c; }
    else { r = c; g = 0; b = x; }

    ColorVec rgb = {r + m, g + m, b + m};
    rgb.mult(255);
    rgb.restrictRGB();

    return rgb;
}

__host__ __device__ ColorVec Color3D::x255(ColorVec vec) {
    return ColorVec{vec.v1 * 255, vec.v2 * 255, vec.v3 * 255};
}