#include "hip/hip_runtime.h"
#include <SFMLTexture.cuh>
#include <CsLogHandle.h>

// Playground
#include <Wall.cuh>
#include <Cube3D.cuh>

int main() {

    // =================== INITIALIZATION =======================
    // Pixel size 4 is the sweet spot for performance and quality

    FpsHandle *FPS = new FpsHandle();
    Camera3D *CAM = new Camera3D();
    Render3D *RENDER = new Render3D(CAM, 1600, 900, 4);
    SFMLTexture *TEXTURE = new SFMLTexture(RENDER);

    // Debugging
    CsLogHandle *CSLOG = new CsLogHandle();

    sf::RenderWindow WINDOW(
        sf::VideoMode(RENDER->W_WIDTH, RENDER->W_HEIGHT), RENDER->W_TITLE
    );
    WINDOW.setMouseCursorVisible(false);

    // =================== EXPERIMENTATION =======================

    // Initialize stuff
    CAM->pos = Vec3D(0, 90, -120);
    CAM->ang = Vec3D(0, 0, 0);

    RENDER->DEFAULT_COLOR = Color3D(0, 0, 0);
    RENDER->LIGHT.pos = Vec3D(75, 140, 75);

    std::vector<Tri3D> TRI_VEC;

    std::vector<Tri3D> MODEL_1 = Tri3D::readObj(
        "assets/Models/Sukuna.obj"
    );
    for (int i = 0; i < MODEL_1.size(); i++) {
        MODEL_1[i].scale(Vec3D(), Vec3D(10, 10, 10));
        MODEL_1[i].color = Color3D(255, 255, 255);
        MODEL_1[i].isTwoSided = true;
        TRI_VEC.push_back(MODEL_1[i]);
    }

    // Function y = f(x, z) to create a 3D graph
    std::vector<std::vector<Vec3D>> points;
    for (double x = -10; x < 10; x += 0.1) {
        points.push_back(std::vector<Vec3D>());
        for (double z = -10; z < 10; z += 0.1) {
            double y = 0;

            points.back().push_back(Vec3D(x, y, z));
        }
    }

    for (size_t x = 0; x < points.size() - 1; x++) {
        for (size_t z = 0; z < points[x].size() - 1; z++) {
            double cx = 50 + 150 * double(x) / points.size();
            double cz = 50 + 150 * double(z) / points[x].size();
            double csqrt = 255 * sqrt((cx*cx + cz*cz) / 65025);
            Color3D color = Color3D(232, 211, 139 + 20 * double(x) / points.size());
            color = Color3D(255, 255, 255);

            Tri3D tri1 = Tri3D(
                points[x][z], points[x + 1][z], points[x][z + 1],
                color
            );
            Tri3D tri2 = Tri3D(
                points[x][z + 1], points[x + 1][z], points[x + 1][z + 1],
                color
            );

            tri1.scale(Vec3D(), Vec3D(20, 20, 20));
            tri2.scale(Vec3D(), Vec3D(20, 20, 20));

            if (tri1.normal.y < 0) tri1.normal = Vec3D::mult(tri1.normal, -1);
            if (tri2.normal.y < 0) tri2.normal = Vec3D::mult(tri2.normal, -1);

            TRI_VEC.push_back(tri1);
            TRI_VEC.push_back(tri2);
        }
    }

    size_t tri_count = TRI_VEC.size();
    Tri3D *tri_test = new Tri3D[tri_count];
    RENDER->mallocTris(tri_count);

    for (int i = 0; i < tri_count; i++)
        tri_test[i] = TRI_VEC[i];

    // Unrelated stuff
    double rainbowR = 255;
    double rainbowG = 0;
    double rainbowB = 0;
    short cycle = 0;

    while (WINDOW.isOpen()) {
        FPS->startFrame();

        // Resets
        CSLOG->clear();

    // =================== EVENT HANDLING =======================
        sf::Event event;
        while (WINDOW.pollEvent(event)) {
            if (event.type == sf::Event::Closed ||
                sf::Keyboard::isKeyPressed(sf::Keyboard::Escape)) {
                WINDOW.close();
            }

            // Scroll to change fov 
            if (event.type == sf::Event::MouseWheelScrolled) {
                if (event.mouseWheelScroll.delta > 0)
                    CAM->fov -= 5; // Zoom in
                else
                    CAM->fov += 5; // Zoom out

                CAM->fov = std::max(10.0, std::min(CAM->fov, 170.0));
            }

            // Press space to face (0, 0, 0)
            if (sf::Keyboard::isKeyPressed(sf::Keyboard::Space)) {
                CAM->facePoint(Vec3D(0, 0, 0));
            }

            // F1 to toggle focus
            if (sf::Keyboard::isKeyPressed(sf::Keyboard::F1)) {
                CAM->focus = !CAM->focus;
                WINDOW.setMouseCursorVisible(!CAM->focus);
            }
        }

        if (CAM->focus) {
            // Mouse movement handling
            sf::Vector2i mousePos = sf::Mouse::getPosition(WINDOW);
            sf::Mouse::setPosition(sf::Vector2i(RENDER->W_CENTER_X, RENDER->W_CENTER_Y), WINDOW);

            // Move from center
            int dMx = mousePos.x - RENDER->W_CENTER_X;
            int dMy = mousePos.y - RENDER->W_CENTER_Y;

            // Camera look around
            CAM->ang.x -= dMy * CAM->m_sens * FPS->dTimeSec;
            CAM->ang.y += dMx * CAM->m_sens * FPS->dTimeSec;

            // Restrict the angle
            CAM->ang.x = std::max(-M_PI_2, std::min(M_PI_2, CAM->ang.x));

            if (CAM->ang.y > M_2PI) CAM->ang.y -= M_2PI;
            if (CAM->ang.y < 0) CAM->ang.y += M_2PI;

            bool m_left = sf::Mouse::isButtonPressed(sf::Mouse::Left);
            bool m_right = sf::Mouse::isButtonPressed(sf::Mouse::Right);

            // Mouse Click = move forward
            if (m_left && !m_right)      CAM->vel = 1;
            else if (m_right && !m_left) CAM->vel = -1;
            else                         CAM->vel = 0;
        }
        CAM->update();

    // ============== WHERE THE REAL FUN BEGINS =====================   

        // =================== LOGGING =======================

        // Rainbow color
        double step = 120 * FPS->dTimeSec;
        if (cycle == 0) {
            rainbowG += step; rainbowR -= step;
            if (rainbowG >= 255) cycle = 1;
        } else if (cycle == 1) {
            rainbowB += step; rainbowG -= step;
            if (rainbowB >= 255) cycle = 2;
        } else if (cycle == 2) {
            rainbowR += step; rainbowB -= step;
            if (rainbowR >= 255) cycle = 0;
        }

        sf::Color rainbow = sf::Color(rainbowR, rainbowG, rainbowB);
        CSLOG->addLog("Welcome to AsczEngine 2.0", rainbow, 1);

        // FPS <= 10: Fully Red
        // FPS >= 60: Fully Green
        double gRatio = double(FPS->fps - 10) / 50;
        gRatio = std::max(0.0, std::min(gRatio, 1.0));
        sf::Color fpsColor((1 - gRatio) * 255, gRatio * 255, 0);
        CSLOG->addLog("FPS: " + std::to_string(FPS->fps), fpsColor);

        std::string add = ""; // For million and billion
        size_t displayNum = tri_count;
        if (displayNum > 1'000'000'000) {
            displayNum /= 1'000'000'000; add = "B";
        } else if (displayNum > 1'000'000) {
            displayNum /= 1'000'000; add = "M";
        }
        CSLOG->addLog("TRI_COUNT: " + std::to_string(displayNum) + add, sf::Color::Yellow);

        CSLOG->addLog(CAM->log, sf::Color::Red);

        // ================= Playground ====================

        // Rotate the light source
        RENDER->LIGHT.pos = Vec3D::rotate(
            RENDER->LIGHT.pos, Vec3D(0, 0, 0),
            // Vec3D(M_PI / 6 * FPS->dTimeSec, 0, M_PI / 6 * FPS->dTimeSec)
            Vec3D(0, M_PI / 6 * FPS->dTimeSec, 0)
        );

        // // YOU are the light source
        // RENDER->LIGHT.pos = CAM->pos;
        // RENDER->LIGHT.normal = CAM->plane.normal;

        // ======= Main graphic rendering pipeline =======
        RENDER->renderGPU(tri_test, tri_count);

        TEXTURE->updateTexture(RENDER);
        WINDOW.draw(TEXTURE->sprite);
        CSLOG->drawLog(WINDOW);
        WINDOW.display();

        FPS->endFrame();
    }

    delete CAM;
    delete FPS;
    delete CSLOG;
    delete RENDER;
    delete TEXTURE;
    delete[] tri_test;

    return 0;
}