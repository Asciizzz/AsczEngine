#include "hip/hip_runtime.h"
#include <SFMLTexture.cuh>
#include <CsLogHandle.h>

// Playground
#include <Wall.cuh>
#include <Cube3D.cuh>

int main() {
    FpsHandle *FPS = new FpsHandle();
    Camera3D *CAM = new Camera3D();
    Render3D *RENDER = new Render3D(CAM);
    SFMLTexture *TEXTURE = new SFMLTexture(RENDER);
    CAM->w_center_x = RENDER->W_CENTER_X;
    CAM->w_center_y = RENDER->W_CENTER_Y;

    CAM->pos = Vec3D(0, 20, 0);

    // Debugging
    CsLogHandle *CSLOG = new CsLogHandle();

    sf::RenderWindow WINDOW(
        sf::VideoMode(RENDER->W_WIDTH, RENDER->W_HEIGHT), RENDER->W_TITLE
    );
    WINDOW.setMouseCursorVisible(false);

    // =================== EXPERIMENTATION =======================

    // Function y = f(x, z) to create a 3D graph
    std::vector<std::vector<Vec3D>> points;
    std::vector<Tri3D> tris;
    for (double x = -10; x < 10; x += 0.1) {
        points.push_back(std::vector<Vec3D>());
        for (double z = -10; z < 10; z += 0.1) {
            double y = sin(x) * cos(z);
            // double y = 0;

            points.back().push_back(Vec3D(x, y, z));
        }
    }

    for (size_t x = 0; x < points.size() - 1; x++) {
        for (size_t z = 0; z < points[x].size() - 1; z++) {
            double c1 = 50 + 150 * double(x) / points.size();
            double c2 = 50 + 150 * double(z) / points[x].size();

            Color3D color = Color3D(
                c1, 180, c2
            );

            Tri3D tri1 = Tri3D(
                points[x][z], points[x + 1][z], points[x][z + 1],
                color
            );
            Tri3D tri2 = Tri3D(
                points[x][z + 1], points[x + 1][z], points[x + 1][z + 1],
                color
            );

            if (tri1.normal.y < 0) tri1.normal = Vec3D::mult(tri1.normal, -1);
            if (tri2.normal.y < 0) tri2.normal = Vec3D::mult(tri2.normal, -1);

            tris.push_back(tri1);
            tris.push_back(tri2);
        }
    }

    size_t tri_count = tris.size();
    Tri3D *tri_test = new Tri3D[tri_count];

    for (size_t i = 0; i < tri_count; i++) {
        tris[i].v1 = Vec3D::scale(tris[i].v1, Vec3D(), 20);
        tris[i].v2 = Vec3D::scale(tris[i].v2, Vec3D(), 20);
        tris[i].v3 = Vec3D::scale(tris[i].v3, Vec3D(), 20);

        tri_test[i] = tris[i];
    }

    // Unrelated stuff
    double rainbowR = 255;
    double rainbowG = 0;
    double rainbowB = 0;
    short cycle = 0;

    while (WINDOW.isOpen()) {
        // Frame start
        FPS->startFrame();

        // Resets
        CSLOG->clear();
        RENDER->reset();
        WINDOW.clear(sf::Color::White);

    // =================== EVENT HANDLING =======================
        sf::Event event;
        while (WINDOW.pollEvent(event)) {
            if (event.type == sf::Event::Closed ||
                sf::Keyboard::isKeyPressed(sf::Keyboard::Escape)) {
                WINDOW.close();
            }

            // Press space to toggle focus
            if (sf::Keyboard::isKeyPressed(sf::Keyboard::Space)) {
                CAM->focus = !CAM->focus;

                // Hide/unhide cursor
                WINDOW.setMouseCursorVisible(!CAM->focus);
            }
        }

        if (CAM->focus) {
            // Mouse movement handling
            sf::Vector2i mousePos = sf::Mouse::getPosition(WINDOW);
            sf::Mouse::setPosition(sf::Vector2i(RENDER->W_CENTER_X, RENDER->W_CENTER_Y), WINDOW);

            // Move from center
            int dMx = mousePos.x - RENDER->W_CENTER_X;
            int dMy = mousePos.y - RENDER->W_CENTER_Y;

            // Camera look around
            CAM->ang.x -= dMy * CAM->m_sens * FPS->dTimeSec;
            CAM->ang.y += dMx * CAM->m_sens * FPS->dTimeSec;

            // Restrict the angle
            CAM->ang.x = std::max(-M_PI_2, std::min(M_PI_2, CAM->ang.x));

            if (CAM->ang.y > M_2PI) CAM->ang.y -= M_2PI;
            if (CAM->ang.y < 0) CAM->ang.y += M_2PI;

            bool m_left = sf::Mouse::isButtonPressed(sf::Mouse::Left);
            bool m_right = sf::Mouse::isButtonPressed(sf::Mouse::Right);

            // Mouse Click = move forward
            if (m_left && !m_right)      CAM->vel = 1;
            else if (m_right && !m_left) CAM->vel = -1;
            else                         CAM->vel = 0;
        }
        CAM->update();

    // ============== WHERE THE REAL FUN BEGINS =====================   

        // =================== LOGGING =======================

        // Rainbow color
        double step = 120 * FPS->dTimeSec;
        if (cycle == 0) {
            rainbowG += step; rainbowR -= step;
            if (rainbowG >= 255) cycle = 1;
        } else if (cycle == 1) {
            rainbowB += step; rainbowG -= step;
            if (rainbowB >= 255) cycle = 2;
        } else if (cycle == 2) {
            rainbowR += step; rainbowB -= step;
            if (rainbowR >= 255) cycle = 0;
        }
        // Ensure the value is in range
        rainbowR = std::max(0.0, std::min(rainbowR, 255.0));
        rainbowG = std::max(0.0, std::min(rainbowG, 255.0));
        rainbowB = std::max(0.0, std::min(rainbowB, 255.0));

        sf::Color rainbow = sf::Color(rainbowR, rainbowG, rainbowB);
        CSLOG->addLog("Welcome to AsczEngine 2.0", rainbow, 1);

        // FPS <= 10: Fully Red
        // FPS >= 60: Fully Green
        double gRatio = double(FPS->fps - 10) / 50;
        gRatio = std::max(0.0, std::min(gRatio, 1.0));
        sf::Color fpsColor((1 - gRatio) * 255, gRatio * 255, 0);
        CSLOG->addLog("FPS: " + std::to_string(FPS->fps), fpsColor);

        CSLOG->addLog(CAM->log, sf::Color::Cyan);

        // ================= Playground ====================

        // Rotate the light source
        RENDER->light.pos = Vec3D::rotate(
            RENDER->light.pos, Vec3D(0, 0, 0),
            Vec3D(M_PI / 6 * FPS->dTimeSec, 0, M_PI / 6 * FPS->dTimeSec)
        );

        // // YOU are the light source
        // RENDER->light.pos = CAM->pos;

        // ======= Main graphic rendering pipeline =======
        RENDER->renderGPU(tri_test, tri_count);

        // == SFML Rendering that ACTUALLY support parallelism ==
        /*
        Idea: since you cant just execute draw function in parallel, you can
        instead create a texture, fill it with pixels IN PARALLEL, and then
        draw the texture to the window. This way, you can utilize the GPU
        to fill the pixels, and the CPU to draw the texture.

        Jesus Christ, if only SFML has a way to draw pixels in parallel
        Oh wait isnt it related to OpenGL? I think it is...
        */

        // Update the texture
        TEXTURE->updateTexture(RENDER);

        // Draw the texture
        WINDOW.draw(TEXTURE->sprite);

        // Draw the log
        CSLOG->drawLog(WINDOW);

        // Display the window
        WINDOW.display();

        // Frame end
        FPS->endFrame();
    }

    delete CAM;
    delete FPS;
    delete CSLOG;
    delete RENDER;
    delete TEXTURE;
    delete[] tri_test;

    return 0;
}