#include "hip/hip_runtime.h"
#include <SFMLTexture.cuh>
#include <CsLogHandle.h>

// Playground
#include <Wall.cuh>
#include <Cube3D.cuh>

int main() {

    // =================== INITIALIZATION =======================
    FpsHandle *FPS = new FpsHandle();
    Camera3D *CAM = new Camera3D();
    Render3D *RENDER = new Render3D(CAM, 1600, 900, 4);
    SFMLTexture *TEXTURE = new SFMLTexture(RENDER);

    // Debugging
    CsLogHandle *CSLOG = new CsLogHandle();

    sf::RenderWindow WINDOW(
        sf::VideoMode(RENDER->W_WIDTH, RENDER->W_HEIGHT), RENDER->W_TITLE
    );
    WINDOW.setMouseCursorVisible(false);

    // =================== EXPERIMENTATION =======================

    // Initialize stuff
    CAM->pos = Vec3D(0, 0, -50);
    CAM->ang = Vec3D(0, M_PI, 0);

    RENDER->LIGHT.pos = Vec3D(0, 0, 12);

    std::vector<Tri3D> TRI_VEC = Tri3D::readObj(
        "assets/Models/Sukuna.obj"
    );

    // Create a cube
    double size = 1.5;

    // Positive X face
    TRI_VEC.push_back(Tri3D(
        Vec3D(size, size, size), Vec3D(size, -size, size), Vec3D(size, -size, -size),
        Vec3D(-1, 0, 0), Color3D(255, 255, 255)
    ));
    TRI_VEC.push_back(Tri3D(
        Vec3D(size, size, size), Vec3D(size, -size, -size), Vec3D(size, size, -size),
        Vec3D(-1, 0, 0), Color3D(255, 255, 255)
    ));
    // Negative X face
    TRI_VEC.push_back(Tri3D(
        Vec3D(-size, size, size), Vec3D(-size, -size, size), Vec3D(-size, -size, -size),
        Vec3D(1, 0, 0), Color3D(255, 255, 255)
    ));
    TRI_VEC.push_back(Tri3D(
        Vec3D(-size, size, size), Vec3D(-size, -size, -size), Vec3D(-size, size, -size),
        Vec3D(1, 0, 0), Color3D(255, 255, 255)
    ));

    // Positive Y face
    TRI_VEC.push_back(Tri3D(
        Vec3D(size, size, size), Vec3D(-size, size, size), Vec3D(-size, size, -size),
        Vec3D(0, -1, 0), Color3D(255, 255, 255)
    ));
    TRI_VEC.push_back(Tri3D(
        Vec3D(size, size, size), Vec3D(-size, size, -size), Vec3D(size, size, -size),
        Vec3D(0, -1, 0), Color3D(255, 255, 255)
    ));
    // Negative Y face
    TRI_VEC.push_back(Tri3D(
        Vec3D(size, -size, size), Vec3D(-size, -size, size), Vec3D(-size, -size, -size),
        Vec3D(0, 1, 0), Color3D(255, 255, 255)
    ));
    TRI_VEC.push_back(Tri3D(
        Vec3D(size, -size, size), Vec3D(-size, -size, -size), Vec3D(size, -size, -size),
        Vec3D(0, 1, 0), Color3D(255, 255, 255)
    ));

    // Positive Z face
    TRI_VEC.push_back(Tri3D(
        Vec3D(size, size, size), Vec3D(-size, size, size), Vec3D(-size, -size, size),
        Vec3D(0, 0, -1), Color3D(255, 255, 255)
    ));
    TRI_VEC.push_back(Tri3D(
        Vec3D(size, size, size), Vec3D(-size, -size, size), Vec3D(size, -size, size),
        Vec3D(0, 0, -1), Color3D(255, 255, 255)
    ));
    // // Negative Z face
    // TRI_VEC.push_back(Tri3D(
    //     Vec3D(size, size, -size), Vec3D(-size, size, -size), Vec3D(-size, -size, -size),
    //     Vec3D(0, 0, 1)
    // ));
    // TRI_VEC.push_back(Tri3D(
    //     Vec3D(size, size, -size), Vec3D(-size, -size, -size), Vec3D(size, -size, -size),
    //     Vec3D(0, 0, 1)
    // ));

    size_t tri_count = TRI_VEC.size();
    tri_count += tri_count % 2;
    size_t tri_chunk = tri_count / 2;
    Tri3D *tri_test = new Tri3D[tri_count];
    Tri3D *tri_part1 = new Tri3D[tri_chunk];
    Tri3D *tri_part2 = new Tri3D[tri_chunk];

    for (int i = 0; i < tri_count; i++) {
        TRI_VEC[i].v1 = Vec3D::scale(TRI_VEC[i].v1, Vec3D(), 10);
        TRI_VEC[i].v2 = Vec3D::scale(TRI_VEC[i].v2, Vec3D(), 10);
        TRI_VEC[i].v3 = Vec3D::scale(TRI_VEC[i].v3, Vec3D(), 10);

        // int rgb = i % 2 == 0 ? 255 : 20;
        // TRI_VEC[i].color = Color3D(rgb, rgb, rgb);
        // TRI_VEC[i].lighting = false;

        tri_test[i] = TRI_VEC[i];
    }

    for (int i = 0; i < tri_chunk; i++) {
        tri_part1[i] = tri_test[i];
        tri_part2[i] = tri_test[i + tri_chunk];
    }

    // Unrelated stuff
    double rainbowR = 255;
    double rainbowG = 0;
    double rainbowB = 0;
    short cycle = 0;

    while (WINDOW.isOpen()) {
        FPS->startFrame();

        // Resets
        RENDER->reset();
        CSLOG->clear();

    // =================== EVENT HANDLING =======================
        sf::Event event;
        while (WINDOW.pollEvent(event)) {
            if (event.type == sf::Event::Closed ||
                sf::Keyboard::isKeyPressed(sf::Keyboard::Escape)) {
                WINDOW.close();
            }

            // Scroll to change fov 
            if (event.type == sf::Event::MouseWheelScrolled) {
                if (event.mouseWheelScroll.delta > 0)
                    CAM->fov -= 5; // Zoom in
                else
                    CAM->fov += 5; // Zoom out

                CAM->fov = std::max(10.0, std::min(CAM->fov, 170.0));
            }

            // Press space to face (0, 0, 0)
            if (sf::Keyboard::isKeyPressed(sf::Keyboard::Space)) {
                CAM->facePoint(Vec3D(0, 0, 0));
            }

            // F1 to toggle focus
            if (sf::Keyboard::isKeyPressed(sf::Keyboard::F1)) {
                CAM->focus = !CAM->focus;
                WINDOW.setMouseCursorVisible(!CAM->focus);
            }
        }

        if (CAM->focus) {
            // Mouse movement handling
            sf::Vector2i mousePos = sf::Mouse::getPosition(WINDOW);
            sf::Mouse::setPosition(sf::Vector2i(RENDER->W_CENTER_X, RENDER->W_CENTER_Y), WINDOW);

            // Move from center
            int dMx = mousePos.x - RENDER->W_CENTER_X;
            int dMy = mousePos.y - RENDER->W_CENTER_Y;

            // Camera look around
            CAM->ang.x -= dMy * CAM->m_sens * FPS->dTimeSec;
            CAM->ang.y += dMx * CAM->m_sens * FPS->dTimeSec;

            // Restrict the angle
            CAM->ang.x = std::max(-M_PI_2, std::min(M_PI_2, CAM->ang.x));

            if (CAM->ang.y > M_2PI) CAM->ang.y -= M_2PI;
            if (CAM->ang.y < 0) CAM->ang.y += M_2PI;

            bool m_left = sf::Mouse::isButtonPressed(sf::Mouse::Left);
            bool m_right = sf::Mouse::isButtonPressed(sf::Mouse::Right);

            // Mouse Click = move forward
            if (m_left && !m_right)      CAM->vel = 1;
            else if (m_right && !m_left) CAM->vel = -1;
            else                         CAM->vel = 0;
        }
        CAM->update();

    // ============== WHERE THE REAL FUN BEGINS =====================   

        // =================== LOGGING =======================

        // Rainbow color
        double step = 120 * FPS->dTimeSec;
        if (cycle == 0) {
            rainbowG += step; rainbowR -= step;
            if (rainbowG >= 255) cycle = 1;
        } else if (cycle == 1) {
            rainbowB += step; rainbowG -= step;
            if (rainbowB >= 255) cycle = 2;
        } else if (cycle == 2) {
            rainbowR += step; rainbowB -= step;
            if (rainbowR >= 255) cycle = 0;
        }

        sf::Color rainbow = sf::Color(rainbowR, rainbowG, rainbowB);
        CSLOG->addLog("Welcome to AsczEngine 2.0", rainbow, 1);

        // FPS <= 10: Fully Red
        // FPS >= 60: Fully Green
        double gRatio = double(FPS->fps - 10) / 50;
        gRatio = std::max(0.0, std::min(gRatio, 1.0));
        sf::Color fpsColor((1 - gRatio) * 255, gRatio * 255, 0);
        CSLOG->addLog("FPS: " + std::to_string(FPS->fps), fpsColor);

        CSLOG->addLog(CAM->log, sf::Color::Cyan);

        // ================= Playground ====================

        // Rotate the light source
        RENDER->LIGHT.pos = Vec3D::rotate(
            RENDER->LIGHT.pos, Vec3D(0, 0, 0),
            // Vec3D(M_PI / 6 * FPS->dTimeSec, 0, M_PI / 6 * FPS->dTimeSec)
            Vec3D(0, M_PI / 6 * FPS->dTimeSec, 0)
        );

        // // YOU are the light source
        // RENDER->LIGHT.pos = CAM->pos;
        // RENDER->LIGHT.normal = CAM->plane.normal;

        // ======= Main graphic rendering pipeline =======
        // RENDER->renderGPU(tri_part1, tri_chunk);
        // RENDER->renderGPU(tri_part2, tri_chunk);
        RENDER->renderGPU(tri_test, tri_count);

        TEXTURE->updateTexture(RENDER);
        WINDOW.draw(TEXTURE->sprite);
        CSLOG->drawLog(WINDOW);
        WINDOW.display();

        FPS->endFrame();
    }

    delete CAM;
    delete FPS;
    delete CSLOG;
    delete RENDER;
    delete TEXTURE;
    delete[] tri_test;

    return 0;
}