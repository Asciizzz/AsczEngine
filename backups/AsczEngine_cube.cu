#include "hip/hip_runtime.h"
#include <Render3D.cuh>
#include <CsLogHandle.h>

using namespace sf;

int main() {
    FpsHandle *FPS = new FpsHandle();
    Camera3D *CAM = new Camera3D();
    Render3D *RENDER = new Render3D(CAM);
    CAM->w_center_x = RENDER->W_CENTER_X;
    CAM->w_center_y = RENDER->W_CENTER_Y;

    // Debugging
    CsLogHandle *CSLOG = new CsLogHandle();

    RenderWindow WINDOW(
        VideoMode(RENDER->W_WIDTH, RENDER->W_HEIGHT), RENDER->W_TITLE
    );
    WINDOW.setMouseCursorVisible(false);

    // =================== EXPERIMENTATION =======================

    // Create a cube
    size_t tri_count = 12;
    Tri3D *tri_test = new Tri3D[tri_count];

    int size = 10;

    // Positive X face
    tri_test[0] = Tri3D(
        Vec3D(size + 0.1, size, size), Vec3D(size + 0.1, -size, size), Vec3D(size + 0.1, -size, -size),
        Vec3D(1, 0, 0), Color3D(255, 0, 0)
    );
    tri_test[1] = Tri3D(
        Vec3D(size + 0.1, size, size), Vec3D(size + 0.1, -size, -size), Vec3D(size + 0.1, size, -size),
        Vec3D(1, 0, 0), Color3D(255, 0, 0)
    );
    // Negative X face
    tri_test[2] = Tri3D(
        Vec3D(-size - 0.1, size, size), Vec3D(-size - 0.1, -size, size), Vec3D(-size - 0.1, -size, -size),
        Vec3D(-1, 0, 0), Color3D(180, 0, 0)
    );
    tri_test[3] = Tri3D(
        Vec3D(-size - 0.1, size, size), Vec3D(-size - 0.1, -size, -size), Vec3D(-size - 0.1, size, -size),
        Vec3D(-1, 0, 0), Color3D(180, 0, 0)
    );

    // Positive Y face
    tri_test[4] = Tri3D(
        Vec3D(size, size + 0.1, size), Vec3D(-size, size + 0.1, size), Vec3D(-size, size + 0.1, -size),
        Vec3D(0, 1, 0), Color3D(0, 255, 0)
    );
    tri_test[5] = Tri3D(
        Vec3D(size, size + 0.1, size), Vec3D(-size, size + 0.1, -size), Vec3D(size, size + 0.1, -size),
        Vec3D(0, 1, 0), Color3D(0, 255, 0)
    );
    // Negative Y face
    tri_test[6] = Tri3D(
        Vec3D(size, -size - 0.1, size), Vec3D(-size, -size - 0.1, size), Vec3D(-size, -size - 0.1, -size),
        Vec3D(0, -1, 0), Color3D(0, 180, 0)
    );
    tri_test[7] = Tri3D(
        Vec3D(size, -size - 0.1, size), Vec3D(-size, -size - 0.1, -size), Vec3D(size, -size - 0.1, -size),
        Vec3D(0, -1, 0), Color3D(0, 180, 0)
    );

    // Positive Z face
    tri_test[8] = Tri3D(
        Vec3D(size, size, size + 0.1), Vec3D(-size, size, size + 0.1), Vec3D(-size, -size, size + 0.1),
        Vec3D(0, 0, 1), Color3D(0, 0, 255)
    );
    tri_test[9] = Tri3D(
        Vec3D(size, size, size + 0.1), Vec3D(-size, -size, size + 0.1), Vec3D(size, -size, size + 0.1),
        Vec3D(0, 0, 1), Color3D(0, 0, 255)
    );
    // Negative Z face
    tri_test[10] = Tri3D(
        Vec3D(size, size, -size - 0.1), Vec3D(-size, size, -size - 0.1), Vec3D(-size, -size, -size - 0.1),
        Vec3D(0, 0, -1), Color3D(0, 0, 180)
    );
    tri_test[11] = Tri3D(
        Vec3D(size, size, -size - 0.1), Vec3D(-size, -size, -size - 0.1), Vec3D(size, -size, -size - 0.1),
        Vec3D(0, 0, -1), Color3D(0, 0, 180)
    );

    // Unrelated stuff
    double rainbowR = 255;
    double rainbowG = 0;
    double rainbowB = 0;
    short cycle = 0;

    while (WINDOW.isOpen()) {
        // Frame start
        FPS->startFrame();

        // =================== EVENT HANDLING =======================
        Event event;
        while (WINDOW.pollEvent(event)) {
            if (event.type == Event::Closed ||
                Keyboard::isKeyPressed(Keyboard::Escape)) {
                WINDOW.close();
            }

            // Press space to toggle focus
            if (Keyboard::isKeyPressed(Keyboard::Space)) {
                CAM->focus = !CAM->focus;

                // Hide/unhide cursor
                WINDOW.setMouseCursorVisible(!CAM->focus);
            }
        }

        if (CAM->focus) {
            // Mouse movement handling
            sf::Vector2i mousePos = sf::Mouse::getPosition(WINDOW);
            sf::Mouse::setPosition(sf::Vector2i(RENDER->W_CENTER_X, RENDER->W_CENTER_Y), WINDOW);

            // Move from center
            int dMx = mousePos.x - RENDER->W_CENTER_X;
            int dMy = mousePos.y - RENDER->W_CENTER_Y;

            // Camera look around
            CAM->ang.x -= dMy * CAM->m_sens * FPS->dTimeSec;
            CAM->ang.y += dMx * CAM->m_sens * FPS->dTimeSec;

            // Restrict the angle
            CAM->ang.x = std::max(-M_PI_2, std::min(M_PI_2, CAM->ang.x));

            if (CAM->ang.y > M_2PI) CAM->ang.y -= M_2PI;
            if (CAM->ang.y < 0) CAM->ang.y += M_2PI;

            // Mouse Click = move forward
            if (Mouse::isButtonPressed(Mouse::Left))       CAM->vel = 1;
            else if (Mouse::isButtonPressed(Mouse::Right)) CAM->vel = -1;
            else                                           CAM->vel = 0;
        }
        CAM->update();

    // ============== WHERE THE REAL FUN BEGINS =====================   

        // =================== LOGGING =======================

        CSLOG->clear();
        // Rainbow color
        double step = 120 * FPS->dTimeSec;
        if (cycle == 0) {
            rainbowG += step; rainbowR -= step;
            if (rainbowG >= 255) cycle = 1;
        } else if (cycle == 1) {
            rainbowB += step; rainbowG -= step;
            if (rainbowB >= 255) cycle = 2;
        } else if (cycle == 2) {
            rainbowR += step; rainbowB -= step;
            if (rainbowR >= 255) cycle = 0;
        }
        // Ensure the value is in range
        rainbowR = std::max(0.0, std::min(rainbowR, 255.0));
        rainbowG = std::max(0.0, std::min(rainbowG, 255.0));
        rainbowB = std::max(0.0, std::min(rainbowB, 255.0));

        sf::Color rainbow = sf::Color(rainbowR, rainbowG, rainbowB);
        CSLOG->addLog("Welcome to AsczEngine 2.0!", rainbow, 1);

        // FPS <= 20: Fully Red
        // FPS >= 80: Fully Green
        double gRatio = double(FPS->fps - 20) / 60;
        gRatio = std::max(0.0, std::min(gRatio, 1.0));
        sf::Color fpsColor((1 - gRatio) * 255, gRatio * 255, 0);
        CSLOG->addLog("FPS: " + std::to_string(FPS->fps), fpsColor);

        // Main rendering
        RENDER->reset();
        RENDER->renderGPU(tri_test, tri_count);

        /* I HAVE TO RANT ABOUT THIS
        I CANT DRAW PIXELS IN PARALLEL
        AS SFML DOESNT SUPPORT MULTITHREADING
        */     
        WINDOW.clear(Color::Black);
        for (int i = 0; i < RENDER->BUFFER_SIZE; i++) {
            RectangleShape pixel(Vector2f(RENDER->PIXEL_SIZE, RENDER->PIXEL_SIZE));
            pixel.setPosition(
                RENDER->BUFFER[i].screen.x * RENDER->PIXEL_SIZE,
                RENDER->BUFFER[i].screen.y * RENDER->PIXEL_SIZE
            );

            // Convert Color3D to sf::Color
            Color3D color = RENDER->BUFFER[i].color;
            pixel.setFillColor(Color(
                color.runtimeRGB.v1, color.runtimeRGB.v2, color.runtimeRGB.v3
            ));
            WINDOW.draw(pixel);
        }

        // Draw the log
        CSLOG->drawLog(WINDOW);

        // Display the window
        WINDOW.display();

        // Frame end
        FPS->endFrame();
    }

    // Clean up
    delete FPS, CAM, RENDER, CSLOG;

    delete[] tri_test;

    return 0;
}